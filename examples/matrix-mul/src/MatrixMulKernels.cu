#include "hip/hip_runtime.h"
#include "MatrixMul.h"
#include <hip/hip_runtime.h>
#include <iostream>

// CPU Matrix Multiplication
void matrixMulCPU(const float* A, const float* B, float* C, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < N; ++k) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

// CUDA kernel for Matrix Multiplication
__global__ void matrixMulKernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void matrixMulGPU(const float* A, const float* B, float* C, int N, int blockX, int blockY) {

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(float));
    hipMalloc((void**)&d_B, N*N*sizeof(float));
    hipMalloc((void**)&d_C, N*N*sizeof(float));


    hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(blockX, blockY);
    dim3 blocksPerGrid((N + blockX - 1) / blockX, (N + blockY - 1) / blockY);
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}